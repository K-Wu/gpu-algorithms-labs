#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdio.h>

#include "template.hu"

#define BLOCK_SIZE 512

// Maximum number of elements that can be inserted into a block queue
#define BQ_CAPACITY 4096

// Number of warp queues per block
#define NUM_WARP_QUEUES 8
// Maximum number of elements that can be inserted into a warp queue
#define WQ_CAPACITY (BQ_CAPACITY / NUM_WARP_QUEUES)

/******************************************************************************
 GPU kernels
*******************************************************************************/

__global__ void gpu_global_queueing_kernel(unsigned int *nodePtrs,
                                           unsigned int *nodeNeighbors,
                                           unsigned int *nodeVisited,
                                           unsigned int *currLevelNodes,
                                           unsigned int *nextLevelNodes,
                                           unsigned int *numCurrLevelNodes,
                                           unsigned int *numNextLevelNodes) {

  // INSERT KERNEL CODE HERE
  // Loop over all nodes in the current level
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  for (unsigned int i = tid; i < *numCurrLevelNodes; i += blockDim.x * gridDim.x) {
    // Loop over all neighbors of the node
    unsigned int node = currLevelNodes[i];
    for (unsigned int j = nodePtrs[node]; j < nodePtrs[node + 1]; j++) {
      unsigned int neighbor = nodeNeighbors[j];
      // If neighbor hasn't been visited yet
      // Mark it as visited
      if (atomicCAS(&nodeVisited[neighbor], 0, 1) == 0) {
        // Add neighbor to global queue
        unsigned int oldIdx    = atomicAdd(numNextLevelNodes, 1);
        nextLevelNodes[oldIdx] = neighbor;
      }
    }
  }
}

__global__ void gpu_block_queueing_kernel(unsigned int *nodePtrs,
                                          unsigned int *nodeNeighbors,
                                          unsigned int *nodeVisited,
                                          unsigned int *currLevelNodes,
                                          unsigned int *nextLevelNodes,
                                          unsigned int *numCurrLevelNodes,
                                          unsigned int *numNextLevelNodes) {
  // INSERT KERNEL CODE HERE

  // Initialize shared memory queue (size should be BQ_CAPACITY)
  __shared__ unsigned int sharedQueue[BQ_CAPACITY];
  __shared__ unsigned int sharedQueueHead;
  __shared__ unsigned int globalQueueCopyBeg;
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (threadIdx.x == 0)
    sharedQueueHead = 0;
  __syncthreads();

  // Loop over all nodes in the current level
  for (unsigned int i = tid; i < *numCurrLevelNodes; i += blockDim.x * gridDim.x) {
    // Loop over all neighbors of the node
    unsigned int node = currLevelNodes[i];
    for (unsigned int j = nodePtrs[node]; j < nodePtrs[node + 1]; j++) {
      unsigned int neighbor = nodeNeighbors[j];
      // If neighbor hasn't been visited yet
      // Mark it as visited
      if (atomicCAS(&nodeVisited[neighbor], 0, 1) == 0) {
        // Add neighbor to block queue
        unsigned int oldIdx = atomicAdd(&sharedQueueHead, 1);
        // If full, add neighbor to global queue
        if (oldIdx >= BQ_CAPACITY) {
          unsigned int oldGlobalIdx    = atomicAdd(numNextLevelNodes, 1);
          nextLevelNodes[oldGlobalIdx] = neighbor;
        } else {
          sharedQueue[oldIdx] = neighbor;
        }
      }
    }
  }
  __syncthreads();

  // Allocate space for block queue to go into global queue
  if (threadIdx.x == 0) {
    if (sharedQueueHead > BQ_CAPACITY) {
      sharedQueueHead = BQ_CAPACITY;
    }
    globalQueueCopyBeg = atomicAdd(numNextLevelNodes, sharedQueueHead);
  }
  __syncthreads();
  // Store block queue in global queue
  for (unsigned int i = threadIdx.x; i < sharedQueueHead; i += blockDim.x) {
    nextLevelNodes[globalQueueCopyBeg + i] = sharedQueue[i];
  }
}

__global__ void gpu_warp_queueing_kernel(unsigned int *nodePtrs,
                                         unsigned int *nodeNeighbors,
                                         unsigned int *nodeVisited,
                                         unsigned int *currLevelNodes,
                                         unsigned int *nextLevelNodes,
                                         unsigned int *numCurrLevelNodes,
                                         unsigned int *numNextLevelNodes) {

  // INSERT KERNEL CODE HERE

  // This version uses NUM_WARP_QUEUES warp queues of capacity
  // WQ_CAPACITY.  Be sure to interleave them as discussed in lecture.
  __shared__ unsigned int warpQueue[WQ_CAPACITY][NUM_WARP_QUEUES];
  __shared__ unsigned int warpQueueHead[NUM_WARP_QUEUES];
  __shared__ unsigned int sharedQueueCopyBeg[NUM_WARP_QUEUES];

  // Don't forget that you also need a block queue of capacity BQ_CAPACITY.
  __shared__ unsigned int sharedQueue[BQ_CAPACITY];
  __shared__ unsigned int sharedQueueHead;
  __shared__ unsigned int globalQueueCopyBeg;

  // Initialize shared memory queues (warp and block)
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (threadIdx.x == 0)
    sharedQueueHead = 0;
  if (threadIdx.x < NUM_WARP_QUEUES) {
    warpQueueHead[threadIdx.x] = 0;
  }
  __syncthreads();

  // Loop over all nodes in the current level
  for (unsigned int i = tid; i < *numCurrLevelNodes; i += blockDim.x * gridDim.x) {
    // Loop over all neighbors of the node
    unsigned int node = currLevelNodes[i];
    for (unsigned int j = nodePtrs[node]; j < nodePtrs[node + 1]; j++) {
      unsigned int neighbor = nodeNeighbors[j];
      // If neighbor hasn't been visited yet
      // Mark it as visited
      if (atomicCAS(&nodeVisited[neighbor], 0, 1) == 0) {
        // Add neighbor to warp queue
        unsigned int oldIdx = atomicAdd(&warpQueueHead[threadIdx.x % 8], 1);
        // If full, add neighbor to block queue
        if (oldIdx >= WQ_CAPACITY) {
          unsigned int oldBlockIdx = atomicAdd(&sharedQueueHead, 1);
          // If full, add neighbor to global queue
          if (oldBlockIdx >= BQ_CAPACITY) {
            unsigned int oldGlobalIdx    = atomicAdd(numNextLevelNodes, 1);
            nextLevelNodes[oldGlobalIdx] = neighbor;
          } else {
            sharedQueue[oldBlockIdx] = neighbor;
          }
        } else {
          warpQueue[oldIdx][threadIdx.x % 8] = neighbor;
        }
      }
    }
  }
  __syncthreads();

  // Allocate space for warp queue to go into block queue
  if (threadIdx.x < NUM_WARP_QUEUES) {
    if (warpQueueHead[threadIdx.x] > WQ_CAPACITY) {
      warpQueueHead[threadIdx.x] = WQ_CAPACITY;
    }
    sharedQueueCopyBeg[threadIdx.x] = atomicAdd(&sharedQueueHead, warpQueueHead[threadIdx.x]);
  }
  __syncthreads();

  // Store warp queues in block queue (use one warp or one thread per queue)
  // Add any nodes that don't fit (remember, space was allocated above)
  //    to the global queue
  for (unsigned int idxWarpQueue = 0; idxWarpQueue < NUM_WARP_QUEUES; idxWarpQueue++) {
    for (unsigned int i = threadIdx.x; i < warpQueueHead[idxWarpQueue]; i += blockDim.x) {
      if (sharedQueueCopyBeg[idxWarpQueue] + i >= BQ_CAPACITY) {
        unsigned int oldGlobalIdx    = atomicAdd(numNextLevelNodes, 1);
        nextLevelNodes[oldGlobalIdx] = warpQueue[i][idxWarpQueue];
      } else {
        sharedQueue[sharedQueueCopyBeg[idxWarpQueue] + i] = warpQueue[i][idxWarpQueue];
      }
    }
  }
  __syncthreads();

  // Saturate block queue counter (too large if warp queues overflowed)
  // Allocate space for block queue to go into global queue
  if (threadIdx.x == 0) {
    if (sharedQueueHead > BQ_CAPACITY) {
      sharedQueueHead = BQ_CAPACITY;
    }
    globalQueueCopyBeg = atomicAdd(numNextLevelNodes, sharedQueueHead);
  }
  __syncthreads();

  // Store block queue in global queue
  for (unsigned int i = threadIdx.x; i < sharedQueueHead; i += blockDim.x) {
    nextLevelNodes[globalQueueCopyBeg + i] = sharedQueue[i];
  }
}

/******************************************************************************
 Functions
*******************************************************************************/
// DON NOT MODIFY THESE FUNCTIONS!

void gpu_global_queueing(unsigned int *nodePtrs, unsigned int *nodeNeighbors, unsigned int *nodeVisited, unsigned int *currLevelNodes,
                         unsigned int *nextLevelNodes, unsigned int *numCurrLevelNodes, unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_global_queueing_kernel<<<numBlocks, BLOCK_SIZE>>>(nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
                                                        numCurrLevelNodes, numNextLevelNodes);
}

void gpu_block_queueing(unsigned int *nodePtrs, unsigned int *nodeNeighbors, unsigned int *nodeVisited, unsigned int *currLevelNodes,
                        unsigned int *nextLevelNodes, unsigned int *numCurrLevelNodes, unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_block_queueing_kernel<<<numBlocks, BLOCK_SIZE>>>(nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
                                                       numCurrLevelNodes, numNextLevelNodes);
}

void gpu_warp_queueing(unsigned int *nodePtrs, unsigned int *nodeNeighbors, unsigned int *nodeVisited, unsigned int *currLevelNodes,
                       unsigned int *nextLevelNodes, unsigned int *numCurrLevelNodes, unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_warp_queueing_kernel<<<numBlocks, BLOCK_SIZE>>>(nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
                                                      numCurrLevelNodes, numNextLevelNodes);
}
